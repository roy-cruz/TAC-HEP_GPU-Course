
#include <hip/hip_runtime.h>
#include <stdio.h>

const int DSIZE_X = 256;
const int DSIZE_Y = 256;
const int NELEMS = DSIZE_X * DSIZE_Y;
const int PRINTSIZE = 3;

__global__ void add_matrix(float *A, float *B, float *C, int width, int height) {

    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    int idy = threadIdx.y + blockDim.y * blockIdx.y;
    int mtrx_idx = idx + width * idy;

    if ((idx < width) && (idy < height))
        C[mtrx_idx] =  A[mtrx_idx] + B[mtrx_idx];
}

__host__ void print_mtrx(const float *arr, const int width, const int height) {
    for (int i = 0; i < width; i++) {
        for (int j = 0; j < height; j++) {
            printf("%f, ", arr[i * width + j]);
        }
        printf("\n");
    }
    printf("\n");
}


int main()
{
   
    // Create and allocate memory for host and device pointers 
    float *h_A, *h_B, *h_C, *d_A, *d_B, *d_C;

    h_A = new float[NELEMS]; 
    h_B = new float[NELEMS];
    h_C = new float[NELEMS];

    for (int i = 0; i < NELEMS; i++) {
        h_A[i] = rand()/(float)RAND_MAX;
        h_B[i] = rand()/(float)RAND_MAX;
        h_C[i] = 0.0f;
    }
    
    hipMalloc(&d_A, NELEMS*sizeof(float));
    hipMalloc(&d_B, NELEMS*sizeof(float));
    hipMalloc(&d_C, NELEMS*sizeof(float));

    // Copy from host to device
    hipMemcpy(d_A, h_A, NELEMS*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, NELEMS*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_C, h_C, NELEMS*sizeof(float), hipMemcpyHostToDevice);

    // Launch the kernel
    dim3 blockSize(16, 16); // 16x16 = 256 threads per block
    dim3 gridSize( // Number of blocks in each dimension
        (DSIZE_X + blockSize.x - 1)/blockSize.x,
        (DSIZE_Y + blockSize.y - 1)/blockSize.y 
    ); 
    
    add_matrix<<<gridSize, blockSize>>>(d_A, d_B, d_C, DSIZE_X, DSIZE_Y);

    // Copy result back to host 
    hipMemcpy(h_C, d_C, NELEMS*sizeof(float), hipMemcpyDeviceToHost);

    // Print and check some elements to make sure the addition was successful
    printf("A:\n");
    print_mtrx(h_A, PRINTSIZE, PRINTSIZE);
    printf("B:\n");
    print_mtrx(h_B, PRINTSIZE, PRINTSIZE);
    printf("==============\n\n");
    printf("C = A + B:\n");
    print_mtrx(h_C, PRINTSIZE, PRINTSIZE);

    // Free the memory
    free(h_A);
    free(h_B);
    free(h_C);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}