
#include <hip/hip_runtime.h>
#include <stdio.h>

const int DSIZE = 40960; // Size of the vectors
const int block_size = 256;
const int grid_size = DSIZE/block_size;

__global__ void swap_vectors(float *A, float *B, const int n_elems) {
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    if (idx < n_elems){
        float temp = A[idx];
        A[idx] = B[idx];
        B[idx] = temp;
    }
}

__host__ void print_array(const float *arr, const int length) {
    for (int i = 0; i < length; i++) {
        printf("%f, ", arr[i]);
    }
    printf("\n");
}


int main() {

    float *h_A, *h_B, *d_A, *d_B;
    // Host vectors
    h_A = new float[DSIZE]; 
    h_B = new float[DSIZE];

    for (int i = 0; i < DSIZE; i++) {
        h_A[i] = rand()/(float)RAND_MAX;
        h_B[i] = rand()/(float)RAND_MAX;
    }

    // Print initial arrays
    printf("Some elements of the arrays before swapping\n");
    printf("A = ");
    print_array(h_A, 4); // Print first 4 elements
    printf("B = ");
    print_array(h_B, 4);

    // Allocate memory for host and device pointers
    hipMalloc(&d_A, DSIZE*sizeof(float)); // d_A is pointer to a device mem address
    hipMalloc(&d_B, DSIZE*sizeof(float));


    // Copy from host to device
    hipMemcpy(d_A, h_A, DSIZE*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, DSIZE*sizeof(float), hipMemcpyHostToDevice);

    // Launch the kernel
    swap_vectors<<<grid_size, block_size>>>(d_A, d_B, DSIZE);

    // Copy back to host 
    hipMemcpy(h_A, d_A, DSIZE*sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(h_B, d_B, DSIZE*sizeof(float), hipMemcpyDeviceToHost);

    // Print and check some elements to make sure swapping was successfull
    printf("Some elements of the arrays after swapping\n");
    printf("A = ");
    print_array(h_A, 4); // Print first 4 elements
    printf("B = ");
    print_array(h_B, 4);

    // Free the memory 
    free(h_A);
    free(h_B);
    hipFree(d_A);
    hipFree(d_B);

    return 0;
}
